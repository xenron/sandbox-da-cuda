#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <cassert>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

cv::Mat imageInputRGBA;
cv::Mat imageOutputRGBA;

uchar4 *d_inputImageRGBA__;
uchar4 *d_outputImageRGBA__;

float *h_filter__;

size_t numRows() { return imageInputRGBA.rows; }
size_t numCols() { return imageInputRGBA.cols; }

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
	if (err != hipSuccess) {
		std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
		std::cerr << hipGetErrorString(err) << " " << func << std::endl;
		exit(1);
	}
}

void preProcess(uchar4 **h_inputImageRGBA, uchar4 **h_outputImageRGBA,
	uchar4 **d_inputImageRGBA, uchar4 **d_outputImageRGBA,
	const std::string &filename) {
	//make sure the context initializes ok
	checkCudaErrors(hipFree(0));

	cv::Mat image = cv::imread(filename.c_str(), CV_LOAD_IMAGE_COLOR);
	if (image.empty()) {
		std::cerr << "Couldn't open file: " << filename << std::endl;
		exit(1);
	}

	cv::cvtColor(image, imageInputRGBA, CV_BGR2RGBA);

	//allocate memory for the output
	imageOutputRGBA.create(image.rows, image.cols, CV_8UC4);

	//This shouldn't ever happen given the way the images are created
	//at least based upon my limited understanding of OpenCV, but better to check
	if (!imageInputRGBA.isContinuous() || !imageOutputRGBA.isContinuous()) {
		std::cerr << "Images aren't continuous!! Exiting." << std::endl;
		exit(1);
	}

	*h_inputImageRGBA = (uchar4 *)imageInputRGBA.ptr<unsigned char>(0);
	*h_outputImageRGBA = (uchar4 *)imageOutputRGBA.ptr<unsigned char>(0);

	const size_t numPixels = numRows() * numCols();
	//allocate memory on the device for both input and output
	checkCudaErrors(hipMalloc(d_inputImageRGBA, sizeof(uchar4) * numPixels));
	checkCudaErrors(hipMalloc(d_outputImageRGBA, sizeof(uchar4) * numPixels));
	checkCudaErrors(hipMemset(*d_outputImageRGBA, 0, numPixels * sizeof(uchar4))); //make sure no memory is left laying around

																					//copy input array to the GPU
	checkCudaErrors(hipMemcpy(*d_inputImageRGBA, *h_inputImageRGBA, sizeof(uchar4) * numPixels, hipMemcpyHostToDevice));

	d_inputImageRGBA__ = *d_inputImageRGBA;
	d_outputImageRGBA__ = *d_outputImageRGBA;

	//now create the filter that they will use

	checkCudaErrors(hipFree(0));

}

//This kernel takes in three color channels and recombines them
//into one image.  The alpha channel is set to 255 to represent
//that this image has no transparency.
__global__
void flipMirror(const uchar4* const inputImageRGBA,
	uchar4* const outputImageRGBA,
	int numRows,
	int numCols)
{
	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;
	const int vertical_thread_1D_pos = (numRows - thread_2D_pos.y) * numCols + thread_2D_pos.x;
	const int horizontal_thread_1D_pos = thread_2D_pos.y * numCols + (numCols - thread_2D_pos.x);

	//make sure we don't try and access memory outside the image
	//by having any threads mapped there return early
	if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
		return;

	outputImageRGBA[vertical_thread_1D_pos].x = inputImageRGBA[thread_1D_pos].x;
	outputImageRGBA[vertical_thread_1D_pos].y = inputImageRGBA[thread_1D_pos].y;
	outputImageRGBA[vertical_thread_1D_pos].z = inputImageRGBA[thread_1D_pos].z;

}

void postProcess(const std::string& output_file, uchar4* data_ptr) {
	cv::Mat output(numRows(), numCols(), CV_8UC4, (void*)data_ptr);
	cv::Mat imageOutputBGR;
	cv::cvtColor(output, imageOutputBGR, CV_RGBA2BGR);
	//output the image
	cv::imwrite(output_file.c_str(), imageOutputBGR);
}

void cleanup() {
	//cleanup
	hipFree(d_inputImageRGBA__);
	hipFree(d_outputImageRGBA__);
	delete[] h_filter__;
}

int main(int argc, char* argv[]) {

	//load input file
	std::string input_file = "cinque_terre_small.jpg";
	//define output file
	std::string output_file = "cinque_terre_small_1.jpg";

	uchar4 *h_inputImageRGBA, *d_inputImageRGBA;
	uchar4 *h_outputImageRGBA, *d_outputImageRGBA;

	//load the image and give us our input and output pointers
	preProcess(&h_inputImageRGBA, &h_outputImageRGBA, &d_inputImageRGBA, &d_outputImageRGBA,
		input_file);

	const dim3 blockSize(16, 16);
	const dim3 gridSize(numCols() / blockSize.x + 1, numRows() / blockSize.y + 1);

	hipDeviceSynchronize(); //checkCudaErrors(hipGetLastError());

	flipMirror << <gridSize, blockSize >> >(d_inputImageRGBA,
		d_outputImageRGBA,
		numRows(),
		numCols());
	hipDeviceSynchronize(); //checkCudaErrors(hipGetLastError());

	size_t numPixels = numRows()*numCols();
	//copy the output back to the host
	checkCudaErrors(hipMemcpy(h_outputImageRGBA, d_outputImageRGBA__, sizeof(uchar4) * numPixels, hipMemcpyDeviceToHost));

	postProcess(output_file, h_outputImageRGBA);

	cleanup();

	return 0;
}

